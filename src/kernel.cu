#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "headers/kernel.h"
#include "headers/vectorcu.cuh"

#include <stdio.h>

__global__ void testKernel() {
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;

	printf("ovr id: %d\n", idx);
}

__global__ void testVectorKernel(const vectorCU<int> vec) {
	printf("%d\n", vec[1]);
}

extern "C" void launchKernel(const unsigned int numBlocks, const unsigned int numThreads) {
	testKernel << <numThreads, numBlocks >> > ();
	hipDeviceSynchronize();
}

extern "C" void testVector() {
	vectorCU<int> vec(1);
	vec.push_back(1);
	vec.push_back(2);
	vec.push_back(3);
	printf("add: %s\n", hipGetErrorString(hipGetLastError()));
	testVectorKernel << <1, 1 >> > (vec);
	printf("kernel: % s\n", hipGetErrorString(hipGetLastError()));
	hipDeviceSynchronize();
}